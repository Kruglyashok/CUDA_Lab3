#include "hip/hip_runtime.h"

#include <iostream>
#include "time.h"
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void printMatr(float *M, int rows, int cols)
{
	if (rows*cols < 17) {
		printf("Matr:\n");
		for (int i = 0; i < rows; ++i) {
			for (int j = 0; j < cols; ++j) {
				printf("%6.2f\t", M[i*cols + j]);
			}
			printf("\n");
		}
	}
}
void matrMult(float* A, float* B, float* C, int rowsA, int colsA, int colsB)
{
	for (int i = 0; i < rowsA; ++i) {
		for (int j = 0; j < colsB; ++j) {
			for (int k = 0; k < colsA; ++k) {
				C[i*colsB + j] += A[i*colsA + k] * B[k*colsB + j];
			}
		}
	}
}
void generate(float* &A, float* &b, int size) {
	srand(time(NULL));
	printf("start generate\n");
	for (int j = 0; j < size; ++j) {
		for (int i = 0; i < size; ++i) {
			//A[i*size + j] = rand() % 100;
			//A[IDX2C(i, j, size)] = i * size + j;
			A[IDX2C(i, j, size)] = rand() % 10;
		}
		b[j] = rand() % 100;
	}
	float* transpA = new float[size];
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			transpA[i*size + j] = A[IDX2C(i, j, size)];
		}
	}
	float* temp = new float[size*size];
	printMatr(A, size, size);
	printMatr(transpA, size, size);
	matrMult(A, transpA, temp, size, size, size);
	//multiplicating matrix on transponed itself generates a
	//positive SEMIdefinite (>= 0)
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			A[IDX2C(i, j, size)] = temp[IDX2C(i, j, size)] / (float)100;
		}
	}
	printMatr(A, size, size);
	printMatr(b, size, 1);
}

int main(int argc, char** argv)
{
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	//int size = atoi(argv[1]);
	//int iters = atoi(argv[2]);
	int size = 4;
	int iters = 1000;
	float *A = new float[size*size], *b = new float[size], *d_A, *d_b, *r, arr, arar, *x, *ar, *x0 = new float[size], t;
	generate(A, b, size);
	printf("finish gen\n");
	for (int i = 0; i < size; ++i) {
		x0[i] = 1;
	}
	printf("filled x0\n");
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		hipblasDestroy(handle);
		printf("CUBLAS initialization failed\n");
		return EXIT_FAILURE;
	}
	printf("start performing malloc\n");
	hipMalloc((void**)&d_A, size*size*sizeof(float));
	hipMalloc((void**)&d_b, size * sizeof(float));
	hipMalloc((void**)&r, size * sizeof(float));
	hipMalloc((void**)&ar, size * sizeof(float));
	hipMalloc((void**)&x, size * sizeof(float));
	printf("performed malloc\n");
	stat = hipblasSetMatrix(size, size, sizeof(*A), A, size, d_A, size);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		hipblasDestroy(handle);
		printf("cublas setmatrix error\n");
		return EXIT_FAILURE;
	}
	stat = hipblasSetVector(size, sizeof(float), b, 1, d_b, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		hipblasDestroy(handle);
		printf("cublas setvector error\n");
		return EXIT_FAILURE;
	}
	stat = hipblasSetVector(size, sizeof(float), x0, 1, x, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		hipblasDestroy(handle);
		printf("cublas setvector error\n");
		return EXIT_FAILURE;
	}
	float alpha = 1.0f;
	float beta = -1.0f;
	float zero = 0.0f;
	float eps = 0.001f;
	bool flag = false;
	printf("begin iters\n");
	int it;
	for (it = 0; it < iters; ++it) {
		stat = hipblasScopy(handle, size, d_b, 1, r, 1); //from d_b into r
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSgemv  error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSgemv(handle, HIPBLAS_OP_N, size, size, &alpha, d_A, size, x, 1, &beta, r, 1);// r = Ax - b; ~ ;r = Ax - r
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSgemv  error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSgemv(handle, HIPBLAS_OP_N, size, size, &alpha, d_A, size, r, 1, &zero, ar, 1);//Ar into ar
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSgemv error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSdot(handle, size, ar, 1, r, 1, &arr); //(Ar,r) 
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSdot error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSdot(handle, size, ar, 1, ar, 1, &arar); //(Ar,Ar) 
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf("hipblasSdot error\n"); 
			hipblasDestroy(handle);
			return EXIT_FAILURE;
		} 
		if (arar == 0) { 
			printf("arar = 0\n");
			hipblasDestroy(handle);
			return EXIT_FAILURE;
		}
		t = - (arr / (float)arar);
		stat = hipblasSaxpy(handle, size, &t, r, 1, x, 1); //x = x - tr
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSaxpy error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSgemv(handle, HIPBLAS_OP_N, size, size, &alpha, d_A, size, x, 1, &zero, r, 1);//Ax into r
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSgemv error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasSaxpy(handle, size, &beta, d_b, 1 ,r, 1); //r = r - d_b
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("hipblasSaxpy error\n");
			return EXIT_FAILURE;
		}
		stat = hipblasGetVector(size, sizeof(float), r, 1, x0, 1);
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			hipblasDestroy(handle);
			printf("cublas get vector");
			hipblasDestroy(handle);
			return EXIT_FAILURE;
		}
		flag = true;
		for (int i = 0; i < size; ++i) {
			if (abs(x0[i]) > eps) { 
				//printf("more than eps %d\n", i);
				flag = false;
			}
		}
		if (flag) break;
	}
	printf("iters: %d\n", it);
	stat = hipblasGetVector(size, sizeof(float), x, 1, x0, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS) {hipblasDestroy(handle);
		printf("cublas get vector");
		hipblasDestroy(handle);
		return EXIT_FAILURE;
	}
	hipFree(d_A);
	hipFree(d_b);
	hipFree(x);
	hipFree(ar);
	hipFree(r);
	hipblasDestroy(handle);
	printf("end iters\n");
	printMatr(x0, size, 1);
	delete[]A; delete[]x0; delete[]b;
	return EXIT_SUCCESS;
}
